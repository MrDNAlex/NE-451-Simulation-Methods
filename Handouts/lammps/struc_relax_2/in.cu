#include "hip/hip_runtime.h"
# ----------------------------------------
# LAMMPS Input Script: Cu (FCC) with EAM
# Goal: Find minimum energy FCC configuration
# ----------------------------------------

# ---- Initialize Simulation ---- 
clear                    # Clear previous simulation settings
units           metal    # Use metal units (Å, eV, ps, g/mol)
dimension       3        # 3D simulation
boundary        p p p    # Periodic boundary conditions in x,y,z
atom_style      atomic   # Atoms only (no charges, bonds, etc.)

# ---- Initialize Simulation Box ----
lattice fcc 3.61         # FCC lattice with lattice constant 3.61 Å (approx Cu experimental value)
region  box block 0 1 0 1 0 1 units lattice   # Define unit box with lattice units
create_box 1 box         # Create simulation box for 1 atom type
create_atoms 1 box       # Fill the box with atoms (according to FCC lattice)
replicate 4 4 4          # Replicate unit cell 4x4x4 times → 256 atoms total

# ---- Define Atom Mass ----
mass 1 63.546                 # Define mass for atom type 1 (Cu)

# ---- Define Interatomic Potential ----
pair_style eam           # Use Embedded Atom Method (EAM) potential
pair_coeff * * Cu_u3.eam # Use provided EAM potential file for Cu (example filename)
neighbor 2.0 bin         # Neighbor list cutoff distance = 2.0 Å
neigh_modify delay 0 every 1 check yes  # Update neighbor list every step

# ---- Define Computations ----
compute eng all pe/atom  # Compute per-atom potential energy
compute eatoms all reduce sum c_eng   # Sum of all atomic energies = total potential energy

# ---- Find Minimum Energy Configuration ----
reset_timestep 0         # Reset timestep counter
fix 1 all box/relax iso 0.0 vmax 0.001   # Relax box dimensions under zero pressure
thermo 10                # Print thermodynamic info every 10 steps
thermo_style custom step pe lx ly lz press c_eatoms  # Output specific quantities
min_style cg             # Use conjugate gradient minimizer
minimize 1e-25 1e-25 5000 10000   # Convergence criteria: energy/force tolerance, max iterations

# ---- Define Global Variables ----
variable N_atoms equal "count(all)"     # Number of atoms
variable total_e equal "c_eatoms"       # Total potential energy
variable length equal "lx"              # Box length (x direction, since FCC cube → lattice constant)
variable coh_e equal "v_total_e/v_N_atoms"  # Cohesive energy per atom

# ---- Print Results ----
print "Total energy (eV) = ${total_e};"
print "Number of atoms = ${N_atoms};"
print "Lattice constant (Angstroms) = ${length};"
print "Cohesive energy (eV) = ${coh_e};"

print "All done!"